#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512 // Size of the vectors

// CUDA kernel for dot product
__global__ void dotProduct(int *a, int *b, int *result, int n) {
    __shared__ int partialSum[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int threadID = threadIdx.x;

    partialSum[threadID] = (tid < n) ? a[tid] * b[tid] : 0;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadID < stride)
            partialSum[threadID] += partialSum[threadID + stride];
        __syncthreads();
    }

    if (threadID == 0)
        atomicAdd(result, partialSum[0]);
}

int main() {
    int host_a[N], host_b[N], host_result = 0, *dev_a, *dev_b, *dev_result;

    for (int i = 0; i < N; i++) {
        host_a[i] = i + 1;
        host_b[i] = i + 2;
    }

    size_t size = N * sizeof(int);
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_result, sizeof(int));

    hipMemcpy(dev_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_result, &host_result, sizeof(int), hipMemcpyHostToDevice);

    dotProduct<<<(N + 255) / 256, 256>>>(dev_a, dev_b, dev_result, N);
    hipMemcpy(&host_result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Dot product: %d\n", host_result);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
}
