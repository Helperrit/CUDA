#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function
__global__ void printWelcomeMessage(int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index

    if (idx < N) {
        printf("Welcome to Parallel Programming from thread %d\n", idx);
    }
}

int main() {
    int N = 10;               // Number of times to print the message
    int threadsPerBlock = 4;  // Number of threads per block

    // Calculate the number of blocks needed
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    printWelcomeMessage<<<blocksPerGrid, threadsPerBlock>>>(N);

    // Synchronize to ensure kernel execution completes
    hipDeviceSynchronize();

    return 0;
}
